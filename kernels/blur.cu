#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>

// 10x10 Gaussian kernel with normalized weights
__device__ const float gaussian_kernel[10][10] = {
    {0.0001f, 0.0002f, 0.0004f, 0.0006f, 0.0008f, 0.0008f, 0.0006f, 0.0004f, 0.0002f, 0.0001f},
    {0.0002f, 0.0004f, 0.0008f, 0.0012f, 0.0016f, 0.0016f, 0.0012f, 0.0008f, 0.0004f, 0.0002f},
    {0.0004f, 0.0008f, 0.0016f, 0.0024f, 0.0032f, 0.0032f, 0.0024f, 0.0016f, 0.0008f, 0.0004f},
    {0.0006f, 0.0012f, 0.0024f, 0.0036f, 0.0048f, 0.0048f, 0.0036f, 0.0024f, 0.0012f, 0.0006f},
    {0.0008f, 0.0016f, 0.0032f, 0.0048f, 0.0064f, 0.0064f, 0.0048f, 0.0032f, 0.0016f, 0.0008f},
    {0.0008f, 0.0016f, 0.0032f, 0.0048f, 0.0064f, 0.0064f, 0.0048f, 0.0032f, 0.0016f, 0.0008f},
    {0.0006f, 0.0012f, 0.0024f, 0.0036f, 0.0048f, 0.0048f, 0.0036f, 0.0024f, 0.0012f, 0.0006f},
    {0.0004f, 0.0008f, 0.0016f, 0.0024f, 0.0032f, 0.0032f, 0.0024f, 0.0016f, 0.0008f, 0.0004f},
    {0.0002f, 0.0004f, 0.0008f, 0.0012f, 0.0016f, 0.0016f, 0.0012f, 0.0008f, 0.0004f, 0.0002f},
    {0.0001f, 0.0002f, 0.0004f, 0.0006f, 0.0008f, 0.0008f, 0.0006f, 0.0004f, 0.0002f, 0.0001f}
};

__global__
void blur_kernel(unsigned char* output, unsigned char* input, int width, int height) {
    const int channels = 3;
    const int kernel_size = 10;
    const int kernel_radius = kernel_size / 2;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        for (int c = 0; c < channels; c++) {
            float blur_value = 0.0f;
            
            // Apply 10x10 gaussian kernel
            for (int i = -kernel_radius; i <= kernel_radius; i++) {
                for (int j = -kernel_radius; j <= kernel_radius; j++) {
                    int cur_row = min(max(row + i, 0), height - 1);
                    int cur_col = min(max(col + j, 0), width - 1);
                    int input_idx = (cur_row * width + cur_col) * channels + c;
                    
                    blur_value += (float)input[input_idx] * gaussian_kernel[i+kernel_radius][j+kernel_radius];
                }
            }
            
            int output_idx = (row * width + col) * channels + c;
            output[output_idx] = (unsigned char)blur_value;
        }
    }
}

// helper function for ceiling unsigned integer division
inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

torch::Tensor blur_image(torch::Tensor image) {
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);

    const auto height = image.size(0);
    const auto width = image.size(1);

    auto result = torch::empty_like(image);

    dim3 threads_per_block(16, 16);     // using 256 threads per block
    dim3 number_of_blocks(cdiv(width, threads_per_block.x),
                         cdiv(height, threads_per_block.y));

    blur_kernel<<<number_of_blocks, threads_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        result.data_ptr<unsigned char>(),
        image.data_ptr<unsigned char>(),
        width,
        height
    );

    // check CUDA error status
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
} 