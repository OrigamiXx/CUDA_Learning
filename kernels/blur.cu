#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>

__device__ const float gaussian_kernel[5][5] = {
    {0.04f, 0.08f, 0.12f, 0.08f, 0.04f},
    {0.08f, 0.16f, 0.24f, 0.16f, 0.08f},
    {0.12f, 0.24f, 0.36f, 0.24f, 0.12f},
    {0.08f, 0.16f, 0.24f, 0.16f, 0.08f},
    {0.04f, 0.08f, 0.12f, 0.08f, 0.04f}
};

__global__
void blur_kernel(unsigned char* output, unsigned char* input, int width, int height) {
    const int channels = 3;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        for (int c = 0; c < channels; c++) {
            float blur_value = 0.0f;
            
            // Apply 5x5 gaussian kernel
            for (int i = -2; i <= 2; i++) {
                for (int j = -2; j <= 2; j++) {
                    int cur_row = min(max(row + i, 0), height - 1);
                    int cur_col = min(max(col + j, 0), width - 1);
                    int input_idx = (cur_row * width + cur_col) * channels + c;
                    
                    blur_value += (float)input[input_idx] * gaussian_kernel[i+2][j+2];
                }
            }
            
            int output_idx = (row * width + col) * channels + c;
            output[output_idx] = (unsigned char)blur_value;
        }
    }
}

// helper function for ceiling unsigned integer division
inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

torch::Tensor blur_image(torch::Tensor image) {
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);

    const auto height = image.size(0);
    const auto width = image.size(1);

    auto result = torch::empty_like(image);

    dim3 threads_per_block(16, 16);     // using 256 threads per block
    dim3 number_of_blocks(cdiv(width, threads_per_block.x),
                         cdiv(height, threads_per_block.y));

    blur_kernel<<<number_of_blocks, threads_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        result.data_ptr<unsigned char>(),
        image.data_ptr<unsigned char>(),
        width,
        height
    );

    // check CUDA error status
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
} 